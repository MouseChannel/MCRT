#include "hip/hip_runtime.h"
#include "Helper/CUDA/CUDAManager.hpp"
#include "Wrapper/Device.hpp"

#include <Wrapper/Buffer.hpp>
#include <Wrapper/SwapChain.hpp>
#include "hip/hip_runtime.h"
#include "hipblas.h"


namespace MCRT {
    void CUDA_Manager::Init() {
        // hipDeviceProp_t deviceProp;
        // int device_count = 0;
        // hipGetDeviceCount(&device_count);
        // hipGetDeviceProperties(&deviceProp, 0);
        // int ret = memcmp((void*)&deviceProp.uuid, Context::Get_Singleton()->get_device()->get_deviceUUID(), VK_UUID_SIZE);

        // if (ret == 0) {
        //     checkCudaErrors(hipSetDevice(current_device));
        //     checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
        //     printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
        //            current_device, deviceProp.name, deviceProp.major,
        //            deviceProp.minor);
        //
        //     return current_device;
        // }
        int aut = 0;
    }

    __global__ void real_run(int *device_data) {
        const float freq = 4.0f;
        const size_t stride = gridDim.x * blockDim.x;
        // for (int i = 0; i < blockDim.x; i++) {
        //
        //     device_data[i] = 1234;
        // }
        printf("%d %d %d\n", gridDim.z, blockDim.z,threadIdx.x);
        // Iterate through the entire array in a way that is
        // independent of the grid configuration
        // for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < width * height;
        //      tid += stride) {
        //     // Calculate the x, y coordinates
        //     const size_t y = tid / width;
        //     const size_t x = tid - y * width;
        //     // Normalize x, y to [0,1]
        //     const float u = ((2.0f * x) / width) - 1.0f;
        //     const float v = ((2.0f * y) / height) - 1.0f;
        //     // Calculate the new height value
        //     const float w = 0.5f * sinf(u * freq + time) * cosf(v * freq + time);
        //     // Store this new height value
        //     heightMap[tid] = w;
        //      }
    }

    void CUDA_Manager::Run(void *device_data) {
        real_run<<<1, 8>>>((int *) device_data);
        return;
#define M 3
#define N 3
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float A[9], *InvA;
//        A = (float *) malloc(M * N * sizeof(float));
        InvA = (float *) malloc(M * N * sizeof(float));
        // 初始化矩阵 A
        for (int i = 0; i < M; i++) {
            for (int j = 0; j < N; j++) {
                A[i * N + j] = i + j;
            }
        }

        // 将矩阵 A 和 InvA 复制到 GPU 内存中
        float *d_A, *d_InvA;
        int *info;
        hipMalloc((void **) &d_A, M * N * 4);
        hipMalloc((void **) &d_InvA, M * N * sizeof(float));
        hipMalloc((void **) &info, 4);
//        if (hipMalloc((void **) &info, 4) != hipSuccess) {
//            throw std::runtime_error("errint");
//        }
        if (hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice) != hipSuccess) {
            throw std::runtime_error("err3");
        }
        if (hipMemcpy(A, d_A, M * N * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost) != hipSuccess) {
            throw std::runtime_error("err4");
        }
        auto before =  hipMemcpy(InvA, d_InvA, M * N * 4, hipMemcpyKind::hipMemcpyDeviceToHost);


        auto res = cublasSmatinvBatched(handle, N, &d_A, N, &d_InvA, N, info, 1);
        auto after =  hipMemcpy(InvA, d_InvA, M * N * 4, hipMemcpyKind::hipMemcpyDeviceToHost);

        float data[9] = {1, 1, 1, 1, 1, 1, 1, 1, 1};
        void *host_data = (void *) data;
        int a = 0;
        int r = 0;
        std::cout << "mocheng" << std::endl;
        int host_int[1];
//        auto copy_int = hipMemcpy(A, d_A, 4, hipMemcpyDeviceToHost);

        auto rr = *host_int;

        auto res1 = hipMemcpy(InvA, d_InvA, M * N * 4, hipMemcpyKind::hipMemcpyDeviceToHost);

//        // 将结果矩阵 InvA 从 GPU 内存中复制回主机内存中

//        hipblasGetMatrix(M, N, sizeof(double), d_InvA, M, InvA, M);
//
//       
        std::cout << info << "😘" << res << std::endl;
    }

    void CUDA_Manager::CreateVKExternalSamephore() {

        waitSemaphore.reset(new Semaphore(true));
        signalSemaphore.reset(new Semaphore(true));

    }


    void CUDA_Manager::ImportCudaExternalSemaphore() {
        hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc = {};
        externalSemaphoreHandleDesc.type =
                hipExternalSemaphoreHandleTypeOpaqueFd;
        externalSemaphoreHandleDesc.handle.fd =
                (int) (uintptr_t) GetSemaphoreHandle(waitSemaphore);
        auto res = hipImportExternalSemaphore(&m_cudaWaitSemaphore, &externalSemaphoreHandleDesc);
        if (res != hipSuccess) {
            throw std::runtime_error("fail ImportCudaExternalSemaphore");
        };
        //////////////////////////
        externalSemaphoreHandleDesc.handle.fd =
                (int) (uintptr_t) GetSemaphoreHandle(signalSemaphore);
        res = hipImportExternalSemaphore(&m_cudaSignalSemaphore, &externalSemaphoreHandleDesc);
        if (res != hipSuccess) {
            throw std::runtime_error("fail ImportCudaExternalSemaphore");
        };
    }

    void *CUDA_Manager::GetSemaphoreHandle(std::shared_ptr<Semaphore> &semaphore) {
        // int fd;
        vk::SemaphoreGetFdInfoKHR semaphore_get_fd_info;
        semaphore_get_fd_info.setHandleType(vk::ExternalSemaphoreHandleTypeFlagBits::eOpaqueFd)
                .setSemaphore(semaphore->get_handle());
        auto m_device = Context::Get_Singleton()->get_device()->get_handle();

        int fd = m_device.getSemaphoreFdKHR(semaphore_get_fd_info);
        return (void *) (uintptr_t) fd;
    }

    void *CUDA_Manager::GetMemHandle(std::shared_ptr<Buffer> buffer) {
        auto memoryGetFdInfoKHR = vk::MemoryGetFdInfoKHR()
                .setMemory(buffer->GetMemory())
                .setHandleType(vk::ExternalMemoryHandleTypeFlagBitsKHR::eOpaqueFd);
        auto fd = Context::Get_Singleton()->get_device()->get_handle().getMemoryFdKHR(memoryGetFdInfoKHR);
        return (void *) (uintptr_t) fd;

    }

    void CUDA_Manager::ImportCudaExternalMemory(void **cudaPtr, hipExternalMemory_t &cudaMem,
                                                const std::shared_ptr<Buffer> &buffer) {
        hipExternalMemoryHandleDesc externalMemoryHandleDesc = {};
        externalMemoryHandleDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
        externalMemoryHandleDesc.size = buffer->GetSize();
        externalMemoryHandleDesc.handle.fd =
                (int) (uintptr_t) GetMemHandle(buffer);
        if (hipImportExternalMemory(&cudaMem,
                                     &externalMemoryHandleDesc) != hipSuccess) {
            throw std::runtime_error("err hipImportExternalMemory");
        }
        hipExternalMemoryBufferDesc externalMemBufferDesc = {};
        externalMemBufferDesc.offset = 0;
        externalMemBufferDesc.size = buffer->GetSize();
        externalMemBufferDesc.flags = 0;
        if (hipExternalMemoryGetMappedBuffer(cudaPtr,
                                              cudaMem,
                                              &externalMemBufferDesc) != hipSuccess) {
            throw std::runtime_error("err hipExternalMemoryGetMappedBuffer");
        }

    }


}

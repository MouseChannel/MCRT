#include "hip/hip_runtime.h"
#include "Helper/CUDA/CUDAManager.hpp"
#include "Wrapper/Device.hpp"

#include <Wrapper/Buffer.hpp>
#include <Wrapper/SwapChain.hpp>


namespace MCRT {
void CUDA_Manager::Init()
{
    // hipDeviceProp_t deviceProp;
    // int device_count = 0;
    // hipGetDeviceCount(&device_count);
    // hipGetDeviceProperties(&deviceProp, 0);
    // int ret = memcmp((void*)&deviceProp.uuid, Context::Get_Singleton()->get_device()->get_deviceUUID(), VK_UUID_SIZE);

    // if (ret == 0) {
    //     checkCudaErrors(hipSetDevice(current_device));
    //     checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
    //     printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
    //            current_device, deviceProp.name, deviceProp.major,
    //            deviceProp.minor);
    //
    //     return current_device;
    // }
    int aut = 0;
}

__global__ void real_run(int* device_data)
{
    const float freq = 4.0f;
    const size_t stride = gridDim.x * blockDim.x;
    for (int i = 0; i < blockDim.x; i++) {

        device_data[i] = 1234;
    }
    printf("%d %d \n", gridDim.x, blockDim.y);
    // Iterate through the entire array in a way that is
    // independent of the grid configuration
    // for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < width * height;
    //      tid += stride) {
    //     // Calculate the x, y coordinates
    //     const size_t y = tid / width;
    //     const size_t x = tid - y * width;
    //     // Normalize x, y to [0,1]
    //     const float u = ((2.0f * x) / width) - 1.0f;
    //     const float v = ((2.0f * y) / height) - 1.0f;
    //     // Calculate the new height value
    //     const float w = 0.5f * sinf(u * freq + time) * cosf(v * freq + time);
    //     // Store this new height value
    //     heightMap[tid] = w;
    //      }
}

void CUDA_Manager::Run(void* device_data)
{
    real_run<<<1,10>>>((int*)device_data);
}

void CUDA_Manager::CreateVKExternalSamephore()
{

    waitSemaphore.reset(new Semaphore(true));
    signalSemaphore.reset(new Semaphore(true));

}


void CUDA_Manager::ImportCudaExternalSemaphore()
{
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc = {};
    externalSemaphoreHandleDesc.type =
        hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd =
        (int)(uintptr_t)GetSemaphoreHandle(waitSemaphore);
    auto res = hipImportExternalSemaphore(&m_cudaWaitSemaphore, &externalSemaphoreHandleDesc);
    if (res != hipSuccess) {
        throw std::runtime_error("fail ImportCudaExternalSemaphore");
    };
    //////////////////////////
    externalSemaphoreHandleDesc.handle.fd =
        (int)(uintptr_t)GetSemaphoreHandle(signalSemaphore);
    res = hipImportExternalSemaphore(&m_cudaSignalSemaphore, &externalSemaphoreHandleDesc);
    if (res != hipSuccess) {
        throw std::runtime_error("fail ImportCudaExternalSemaphore");
    };
}

void* CUDA_Manager::GetSemaphoreHandle(std::shared_ptr<Semaphore>& semaphore)
{
    // int fd;
    vk::SemaphoreGetFdInfoKHR semaphore_get_fd_info;
    semaphore_get_fd_info.setHandleType(vk::ExternalSemaphoreHandleTypeFlagBits::eOpaqueFd)
                         .setSemaphore(semaphore->get_handle());
    auto m_device = Context::Get_Singleton()->get_device()->get_handle();

    int fd = m_device.getSemaphoreFdKHR(semaphore_get_fd_info);
    return (void*)(uintptr_t)fd;
}

void* CUDA_Manager::GetMemHandle(std::shared_ptr<Buffer> buffer)
{
    auto memoryGetFdInfoKHR = vk::MemoryGetFdInfoKHR()
                              .setMemory(buffer->GetMemory())
                              .setHandleType(vk::ExternalMemoryHandleTypeFlagBitsKHR::eOpaqueFd);
    auto fd = Context::Get_Singleton()->get_device()->get_handle().getMemoryFdKHR(memoryGetFdInfoKHR);
    return (void*)(uintptr_t)fd;

}

void CUDA_Manager::ImportCudaExternalMemory(void** cudaPtr, hipExternalMemory_t& cudaMem, const std::shared_ptr<Buffer>& buffer)
{
    hipExternalMemoryHandleDesc externalMemoryHandleDesc = {};
    externalMemoryHandleDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    externalMemoryHandleDesc.size = buffer->GetSize();
    externalMemoryHandleDesc.handle.fd =
        (int)(uintptr_t)GetMemHandle(buffer);
    if (hipImportExternalMemory(&cudaMem,
                                 &externalMemoryHandleDesc) != hipSuccess) {
        throw std::runtime_error("err hipImportExternalMemory");
    }
    hipExternalMemoryBufferDesc externalMemBufferDesc = {};
    externalMemBufferDesc.offset = 0;
    externalMemBufferDesc.size = buffer->GetSize();
    externalMemBufferDesc.flags = 0;
    if (hipExternalMemoryGetMappedBuffer(cudaPtr,
                                          cudaMem,
                                          &externalMemBufferDesc) != hipSuccess) {
        throw std::runtime_error("err hipExternalMemoryGetMappedBuffer");
    }

}


}
